// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// gpublas example 2.18 
// the final argument argv[5] controls the use of Tensor Cores.
// 
// RTX 2070
// C:\bin\blasmult.exe 1024 1024 1024 0/1
// A A 1024 x 1024 B 1024 x 1024 gpu time 0.418 ms GFlops 5133.6 GBytes 30801.3 (no TC)
// A 1024 x 1024 B 1024 x 1024 gpu time 0.220 ms GFlops 9741.4 GBytes 58448.3 (with TC)
// 
// RTX 3080
// C:\bin\blasmult.exe 1024 1024 1024 0/1
// A 1024 x 1024 B 1024 x 1024 gpu time 0.168 ms GFlops 12777.6 GBytes 76665.7 (no TC)
// A 1024 x 1024 B 1024 x 1024 gpu time 0.092 ms GFlops 23221.6 GBytes 139329.4 (with TC)

#include "../include/cx.h"
#include "cxtimers.h"
#include <random>
#include "hipblas.h"

int main(int argc,char *argv[])
{
	int Arow = (argc > 1) ? atoi(argv[1]) : 1 << 10; // default 2^10
	int Acol = (argc > 2) ? atoi(argv[2]) : Arow;
	int Brow = Acol;
	int Bcol = (argc > 3) ? atoi(argv[3]) : Brow;
	int Crow = Arow;
	int Ccol = Bcol;
	int useTC = (argc > 4) ? atoi(argv[4]) : 1;   // use TC by default
	int nacc = (argc > 5) ? atoi(argv[5]) : 100;  // for timing

	//printf("params %d %d %d to %d nacc %d\n", Arow, Acol, Bcol, useTC, nacc);

	thrust::host_vector<float>       A(Arow*Acol);
	thrust::host_vector<float>       B(Brow*Bcol);
	thrust::host_vector<float>       C(Crow*Ccol);
	thrust::device_vector<float> dev_A(Arow*Acol);
	thrust::device_vector<float> dev_B(Brow*Bcol);
	thrust::device_vector<float> dev_C(Crow*Ccol);
	thrust::device_vector<float> dev_D(Crow*Ccol);

	// initialise x with random numbers and copy to dx.
	std::default_random_engine gen(12345678);
	std::uniform_real_distribution<float> fran(0.0,1.0);
	for(int k = 0; k<Arow*Acol; k++) A[k] = fran(gen);
	for(int k = 0; k<Brow*Bcol; k++) B[k] = fran(gen);
	for(int k = 0; k<Crow*Ccol; k++) C[k] = 0.0f;

	dev_A = A;  // H2D copy
	dev_B = B;  // H2D copy
	dev_C = C;  // clear

	float alpha = 1.0f; // 128th root of 10
	float beta  = 1.0f;
	hipblasHandle_t handle; 	hipblasCreate(&handle);
	if(useTC != 0) hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH); // optional enable tensor cores
	cx::timer tim;
	for(int k=0;k<nacc;k++){  // C = alpha*(A*B) + beta*C
		hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,Crow,Ccol,Acol,&alpha,dev_A.data().get(),Arow,dev_B.data().get(),Brow,&beta,dev_C.data().get(),Crow);
	}
	beta = 0.0f;  // D = transpose(C)
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,Crow,Ccol,&alpha,dev_C.data().get(),Crow,&beta,dev_C.data().get(),Crow,dev_D.data().get(),Ccol);
	hipDeviceSynchronize();
	double t3 = tim.lap_ms()/(double)(nacc);
	C = dev_D; // D2H copy

	double flops = 2.0*(double)Arow*(double)Acol*(double)Bcol;
	double gflops = flops/(t3*1000000.0);
	double gbytes = gflops*6.0; // i.e 12 bytes per term
	if(useTC==0) printf("A %d x %d B %d x %d gpu time %.3f ms GFlops %.1f GBytes %.1f (no TC)\n",Arow,Acol,Brow,Bcol,t3,gflops,gbytes);
	else         printf("A %d x %d B %d x %d gpu time %.3f ms GFlops %.1f GBytes %.1f (with TC)\n",Arow,Acol,Brow,Bcol,t3,gflops,gbytes);
	return 0;
}
