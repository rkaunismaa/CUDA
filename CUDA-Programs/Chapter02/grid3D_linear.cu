#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// grid3D_linear example 2.4
// 
// RTX 2070
// C:\bin\grid3d_linear.exe 1234567 288 256
// array size   512 x 512 x 256 = 67108864
// thread block 256
// thread  grid 288
// total number of threads in grid 73728
// a[4][363][135] = 1234567 and b[4][363][135] = 1111.110718
// rank_in_block = 135 rank_in_grid = 54919 rank of block_rank_in_grid = 214 pass 16
// 
// RTX 3080
// C:\bin\grid3D_linear.exe 1234567 288 256
// array size   512 x 512 x 256 = 67108864
// thread block 256
// thread  grid 288
// total number of threads in grid 73728
// a[4][363][135] = 1234567 and b[4][363][135] = 1111.110718
// rank_in_block = 135 rank_in_grid = 54919 pass 16 tid offset 1179648

#include "cx.h"

__device__  int   a[256][512][512];  // file scope
__device__  float b[256][512][512];  // file scope

__global__ void grid3D_linear(int nx,int ny,int nz,int id)
{
	int tid = blockIdx.x*blockDim.x+threadIdx.x;

	int array_size = nx*ny*nz;
	int total_threads = gridDim.x*blockDim.x;
	int tid_start = tid;
	int pass = 0;
	while(tid < array_size){
		int x =  tid%nx;        
		int y = (tid/nx)%ny;   
		int z =  tid/(nx*ny); 
		// do some work here
		a[z][y][x] = tid;
		b[z][y][x] = sqrtf((float)a[z][y][x]);
		if(tid == id) {
			printf("array size   %3d x %3d x %3d = %d\n",nx,ny,nz,array_size);
			printf("thread block %3d\n",blockDim.x);
			printf("thread  grid %3d\n",gridDim.x);
			printf("total number of threads in grid %d\n",total_threads);
			printf("a[%d][%d][%d] = %i and b[%d][%d][%d] = %f\n",z,y,x,a[z][y][x],z,y,x,b[z][y][x]);
			printf("rank_in_block = %d rank_in_grid = %d pass %d tid offset %d\n",threadIdx.x,tid_start,pass,tid-tid_start);
		}
		tid += gridDim.x*blockDim.x;
		pass++;
	}
}

int main(int argc,char *argv[])
{
	int id      = (argc > 1) ? atoi(argv[1]) : 12345;
	int blocks  = (argc > 2) ? atoi(argv[2]) : 288;
	int threads = (argc > 3) ? atoi(argv[3]) : 256;
	grid3D_linear<<<blocks,threads>>>(512,512,256,id);
    hipDeviceSynchronize(); // necessary in Linux to see kernel printf
	return 0;
}
