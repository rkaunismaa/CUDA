#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// program gpusum_tla includes:
// example 2.1 tls using while loop
// example 2.2 tla using for   loop
// same code with 5th argument to choose method

#include "../include/cx.h"
#include "cxtimers.h"                    // timers

__host__ __device__ inline float sinsum(float x, int terms)
{
	float x2 = x*x;
	float term = x;   // first term of series
	float sum = term; // sum of terms so far

	for(int n = 1; n < terms; n++){
		term *= -x2 / (2*n*(2*n+1));
		sum += term;
	}

	return sum;

}

// __global__ void gpu_sin(float *sums,int steps,int terms,float step_size) // line 15.1
// {
// 	int step = blockIdx.x*blockDim.x + threadIdx.x; // unique thread ID

// 	if(step<steps){  // line 15.4
// 		float x = step_size * step;
// 		sums[step] = sinsum(x, terms);  // store sin values in array ... line 15.6
// 	}
// }

// Our modiﬁcations to the kernel are changing the if in line 15.4 to while and inserting
// an extra line 15.65 at the end of the while loop. In line 15.65 we increment step by the
// total number of threads in the grid of thread blocks. The while loop will continue until steps
// values have been calculated for all (non-zero) user supplied values of blocks and
// threads. Moreover, and importantly for performance reasons, on each pass through the
// while loop adjacent threads always address adjacent memory locations. Other ways of
// traversing through the data could be devised but the one shown here is the simplest and best.
// This technique of using a while loop with indices having a grid-size stride between passes
// through the loop is called “thread-linear addressing” and is common in CUDA code. It
// should always be considered as an option when porting a loop in host code to CUDA.
__global__ void gpu_sin_tla_whileloop(float *sums, int steps, int terms, float step_size)
{
	int step = blockIdx.x*blockDim.x + threadIdx.x; // start with unique thread ID

	while(step<steps){
		float x = step_size*step;
		sums[step] = sinsum(x, terms);  // save sum
		step += blockDim.x*gridDim.x; //  large stride to next step. ... line 15.65
		// blockDim.x = number of threads in one block
		// gridDim.x = number of blocks in the grid
		// threads = gridDim.x*blockDim.x is total number of threads in launch
	}
}

__global__ void gpu_sin_tla_forloop(float *sums, int steps, int terms, float step_size)
{
	for(int step = blockIdx.x*blockDim.x+threadIdx.x; step<steps; step += gridDim.x*blockDim.x){
		float x = step_size*step;
		sums[step] = sinsum(x, terms);  // save sum
	}
}

int main(int argc,char *argv[])
{
	if (argc < 2) {
		printf("usage gpusum_tla steps|10000000 terms|1000 threads|256 blocks|256 loopkind|1\n");
		return 0;
	}
	
	int steps   = (argc > 1) ? atoi(argv[1])  : 10000000;
	int terms   = (argc > 2) ? atoi(argv[2])  : 1000;
	int threads = (argc > 3) ? atoi(argv[3])  : 256;
	int blocks  = (argc > 4) ? atoi(argv[4])  : (steps+threads-1)/threads;  // ensure threads*blocks >= steps
	int loopkind = (argc > 5) ? atoi(argv[5]) : 1;

	double pi = 3.14159265358979323;
	double step_size = pi / (steps-1); // NB n-1 steps between n points

	thrust::device_vector<float> dsums(steps);         // GPU buffer    
	float *dptr = thrust::raw_pointer_cast(&dsums[0]); // get pointer    

	cx::timer tim;                  // declare and start timer

	if (loopkind== 1) {
		gpu_sin_tla_whileloop<<<blocks,threads>>>(dptr,steps,terms,(float)step_size);  // tla using while loop
	}
	else {
		gpu_sin_tla_forloop<<<blocks,threads>>>(dptr,steps,terms,(float)step_size);  // tla using for loop
	}

	double gpu_sum = thrust::reduce(dsums.begin(),dsums.end());

	double gpu_time = tim.lap_ms(); // get elapsed time

	double rate = (double)steps*(double)terms/(gpu_time*1000000.0);
	gpu_sum -= 0.5*(sinsum(0.0f,terms)+sinsum(pi,terms));
	gpu_sum *= step_size;

	if (loopkind==1)printf("gpu_sum while loop sum = %.10f, steps %d terms %d time %.3f ms config %7d %4d rate %f \n",gpu_sum,steps,terms,gpu_time,blocks,threads,rate);
	else            printf("gpu_sum   for loop sum = %.10f, steps %d terms %d time %.3f ms config %7d %4d rate %f \n",gpu_sum,steps,terms,gpu_time,blocks,threads,rate);

	return 0;
}
