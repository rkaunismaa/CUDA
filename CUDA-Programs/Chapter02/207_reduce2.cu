#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// example 2.7 reduce2
// 
// RTX 2070
// C:\bin\reduce2.exe
// sum of 16777216 numbers: host 8388314.9 14.107 ms GPU 8388314.5 0.166 ms
// 
// RTX 3080
// C:\bin\reduce2.exe 24 256 256
// sum of 16777216 numbers: host 8388314.9 15.662 ms GPU 8388314.5 0.113 ms

#include "cx.h"
#include "cxtimers.h"
#include <random>

__global__ void reduce2(float *y,float *x,int N)
{
	extern __shared__ float tsum[]; // Dynamically Allocated Shared Mem
	int id = threadIdx.x;
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int stride = gridDim.x*blockDim.x;
	tsum[id] = 0.0f;
	for(int k=tid;k<N;k+=stride) tsum[id] += x[k];
	__syncthreads();
	for(int k=blockDim.x/2; k>0; k /= 2){ // power of 2 reduction loop
		if(id<k) tsum[id] += tsum[id+k];
		__syncthreads();
	}
	if(id==0) y[blockIdx.x] = tsum[0]; // store one value per block
}

int main(int argc,char *argv[])
{
	int N       = (argc > 1) ? 1 << atoi(argv[1]) : 1 << 24; // default 2^24
	int blocks  = (argc > 2) ? atoi(argv[2]) : 256;  // power of 2
	int threads = (argc > 3) ? atoi(argv[3]) : 256;
	int nreps   = (argc > 4) ? atoi(argv[4]) : 1000; // set this to 1 for correct answer or >> 1 for timing tests
	thrust::host_vector<float>    x(N);
	thrust::device_vector<float>  dx(N);
	thrust::device_vector<float>  dy(blocks);

	// initialise x with random numbers and copy to dx.
	std::default_random_engine gen(12345678);
	std::uniform_real_distribution<float> fran(0.0,1.0);
	for(int k = 0; k<N; k++) x[k] = fran(gen);
	dx = x;  // H2D copy (N words)
	cx::timer tim;
	double host_sum = 0.0;
	for(int k = 0; k<N; k++) host_sum += x[k]; // host reduce!
	double t1 = tim.lap_ms();

	// simple GPU reduce for any value of N
	tim.reset();
	double gpu_sum = 0.0;
	for(int rep=0;rep<nreps;rep++){
		reduce2<<<blocks,threads,threads*sizeof(float)>>>(dy.data().get(),dx.data().get(),N);
		reduce2<<<     1, blocks, blocks*sizeof(float)>>>(dx.data().get(),dy.data().get(),blocks);
		if(rep==0)  gpu_sum = dx[0];  
	}
	hipDeviceSynchronize();
	double t2 = tim.lap_ms()/nreps;  // time for one pass here
	//double gpu_sum = dx[0]/nreps;          // D2H copy (1 word) 
	printf("sum of %d numbers: host %.1f %.3f ms GPU %.1f %.3f ms\n",N,host_sum,t1,gpu_sum,t2);
	return 0;
}
