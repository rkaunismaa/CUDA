// Created by ChatGPT

#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to multiply matrices A and B
__global__ void matrixMulKernel(float* A, float* B, float* C, int m, int n, int p) {
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < p) {
        float value = 0;
        for (int k = 0; k < n; ++k) {
            value += A[row * n + k] * B[k * p + col];
        }
        C[row * p + col] = value;
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <m> <n> <p>" << std::endl;
        return 1;
    }

    int m = std::atoi(argv[1]);
    int n = std::atoi(argv[2]);
    int p = std::atoi(argv[3]);

    size_t sizeA = m * n * sizeof(float);
    size_t sizeB = n * p * sizeof(float);
    size_t sizeC = m * p * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);

    // Initialize matrices A and B with some values
    for (int i = 0; i < m * n; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < n * p; ++i) {
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((p + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch the kernel
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, p);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize to wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // Print some of the result for verification
    std::cout << "Result matrix C (partial):" << std::endl;
    for (int i = 0; i < std::min(m, 10); ++i) {
        for (int j = 0; j < std::min(p, 10); ++j) {
            std::cout << h_C[i * p + j] << " ";
        }
        std::cout << std::endl;
    }

    // Print the elapsed time
    std::cout << "Time for matrix multiplication: " << elapsedTime << " ms" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
