
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \ 
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int DSIZE = 4096;
//const int DSIZE = 8192;
//const int DSIZE = 16384;
//const int DSIZE = 65536;
//const int DSIZE = 4194304;    // 2 ** 22
//const int DSIZE = 8388608;    // 2 ** 23
// const int DSIZE = 4294967296;  // 2 ** 32
const int block_size = 256;  // CUDA maximum is 1024
// vector add kernel: C = A + B
__global__ void vadd(const float *A, const float *B, float *C, int ds){

  int idx = threadIdx.x + (blockDim.x*blockIdx.x);

  if (idx < ds)
    C[idx] = A[idx] + B[idx];
}

int main(){

  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
  h_A = new float[DSIZE];
  h_B = new float[DSIZE];
  h_C = new float[DSIZE];

  for (int i = 0; i < DSIZE; i++){
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
    h_C[i] = 0;}

  hipMalloc(&d_A, DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");

  hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  //cuda processing sequence step 1 is complete
  vadd<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  //cuda processing sequence step 2 is complete
  hipMemcpy(h_C, d_C, DSIZE*sizeof(float), hipMemcpyDeviceToHost);

  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");

  printf("A[0] = %f\n", h_A[0]);
  printf("B[0] = %f\n", h_B[0]);
  printf("C[0] = %f\n", h_C[0]);

  return 0;
  
}
  
